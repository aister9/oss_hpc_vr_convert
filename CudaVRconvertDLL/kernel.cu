#include "hip/hip_runtime.h"
#include "vrconverter.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void makeMap(uint* maps, int rows, int cols, int eyeWidth);
__global__ void notPersistentMapping(uchar* inputImage, uint* gpuMap, uchar* outputImage, int rows, int cols, int eyeWidth);

extern "C" __declspec(dllexport) void test(char* filepath, char* properties) {
    cv::VideoCapture video;
    video.open(filepath);
    if (!video.isOpened()) {
        return;
    }
    int height = video.get(cv::CAP_PROP_FRAME_HEIGHT);
    int width = video.get(cv::CAP_PROP_FRAME_WIDTH);

    string outputProperties = to_string(width) + "X" + to_string(height);
    strcpy(properties, outputProperties.c_str());
}


extern "C" __declspec(dllexport) void runAVI(char* filepath) {
    cv::VideoCapture video;
    video.open(filepath);
    if (!video.isOpened()) {
        return;
    }

    cv::Mat image;
	cv::Mat outimage;
	int height = video.get(cv::CAP_PROP_FRAME_HEIGHT);
	int width = video.get(cv::CAP_PROP_FRAME_WIDTH);
	uint* maps;
	hipMalloc(&maps, sizeof(uint) * height * width);
	hipMemset(maps, 0, sizeof(uint) * height * width);

	dim3 blocks(32, 32);
	dim3 grid(ceil((float)width / blocks.x), ceil((float)height / blocks.y));

	makeMap << <grid, blocks>> > (maps, height, width, 80);
	hipDeviceSynchronize();

	uchar* gpuInput, *gpuOutput;
	hipMalloc(&gpuInput, sizeof(uchar) * height * width * 3);
	hipMalloc(&gpuOutput, sizeof(uchar) * height * width * 3);

	image = cv::Mat(height, width, CV_8UC3);
	outimage = cv::Mat(height, width, CV_8UC3);
	hipHostMalloc(&image.data, sizeof(uchar) * height * width * 3);
	hipHostMalloc(&outimage.data, sizeof(uchar) * height * width * 3);

    while (cv::waitKey(1) != 27) {
        video >> image;
        if (image.empty()) {
            cv::destroyWindow("images");
            break;
        }
		hipMemcpy(gpuInput, image.data, sizeof(uchar)*height*width*3, hipMemcpyHostToDevice);
		notPersistentMapping << <grid, blocks >> > (gpuInput, maps, gpuOutput, height, width, 80);
		hipDeviceSynchronize();
		hipMemcpy(outimage.data, gpuOutput, sizeof(uchar) * height * width * 3, hipMemcpyDeviceToHost);
        cv::imshow("images", outimage);
    }
}

__global__ void makeMap(uint* maps, int rows, int cols, int eyeWidth) {

	double cx = (double)cols / 2; //width half is center of x
	double cy = (double)rows / 2; //height half is center of y
	double k1 = 0.0, k2 = 0.0;
	if (rows == 2160) {
		k1 = 0.000000014; k2 = 0.000000000000015;
	}
	else if (rows == 1080) {
		k1 = 0.000000037; k2 = 0.00000000000015;
	}
	else if (rows == 4320) {
		k1 = 0.000000007; k2 = 0.0000000000000007;
	}
	//k2 Ŀ������ �ձ׷���, k1 Ŀ������ ���� �����غ�����

	//set index
	int idxX = blockDim.x * blockIdx.x + threadIdx.x;
	int idxY = blockDim.y * blockIdx.y + threadIdx.y;

	if (idxX > cols || idxY > rows) return;

	double rsqaure = pow(cx - idxX, 2) + pow(cy - idxY, 2);
	double hypo = 1 + rsqaure * k1 + pow(rsqaure, 2) * k2; // 1+ kr^2 + kr^4

	int Xd = (idxX - cx) / hypo + cx;
	int Yd = (idxY - cy) / hypo + cy;


	double dCol = (double)cols / (cols/2); // target img 2560*1440, so streoscopic image has 1280*1440
	double dRow = (double)rows / rows; //

	//left img 0 to cols-eyeWidth, right img eyeWidth to cols
	if (Xd < 0 || Yd < 0 || Xd >= cols || Yd >= rows || (idxX % (int)dCol != 0) || (idxY % (int)dRow != 0)) return;
	//idxX % (int)dCol != 0 || idxY % (int)dRow != 0 --> image resize(reduction)

	if (idxX < cols - eyeWidth) {
		//maps[(int)(ny / dRow) * (int)TARGET_WIDTH + (int)(nx / dCol)] = (cols > rows) ? idxX * cols + idxY : idxY * (rows + 1) + idxX;
		maps[(int)(Yd / dRow) * (int)cols + (int)(Xd / dCol)] = (cols > rows) ? idxX * cols + idxY : idxY * (rows + 1) + idxX;
	}
	if (idxX > eyeWidth) {
		//maps[(int)(ny / dRow) * (int)TARGET_WIDTH + (int)(nx / dCol) + (int)TARGET_WIDTH_HALF] = (cols > rows) ? idxX * cols + idxY : idxY * (rows + 1) + idxX;
		maps[(int)(Yd / dRow) * (int)cols + (int)(Xd / dCol) + (int)cols/2] = (cols > rows) ? idxX * cols + idxY : idxY * (rows + 1) + idxX;
	}


	__syncthreads();
}

__global__ void notPersistentMapping(uchar* inputImage, uint* gpuMap, uchar* outputImage, int rows, int cols, int eyeWidth) {
	int idxX = blockDim.x * blockIdx.x + threadIdx.x;
	int idxY = blockDim.y * blockIdx.y + threadIdx.y;

	if (idxX >= cols || idxY >= rows) return;

	int param = (cols > rows) ? cols : rows + 1;

	int dx = 0; int dy = 0;
	dx = gpuMap[idxY * (int)(cols)+idxX] / param;
	dy = gpuMap[idxY * (int)(cols)+idxX] % param;


	if (dy * cols * 3 + dx * 3 >= rows * cols * 3)return;

	if (gpuMap[idxY * (int)(cols)+idxX] != 0) {
		outputImage[idxY * (int)(cols) * 3 + idxX * 3 + 0] = inputImage[dy * (int)(cols) * 3 + dx * 3 + 0];
		outputImage[idxY * (int)(cols) * 3 + idxX * 3 + 1] = inputImage[dy * (int)(cols) * 3 + dx * 3 + 1];
		outputImage[idxY * (int)(cols) * 3 + idxX * 3 + 2] = inputImage[dy * (int)(cols) * 3 + dx * 3 + 2];
	}

	__syncthreads();
}